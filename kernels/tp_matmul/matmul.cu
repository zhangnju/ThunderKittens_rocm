#include "hip/hip_runtime.h"
#include <iostream>
#include <iostream>
#include <random>
#include "kittens.cuh"
#include "prototype.cuh"
#include <hip/hip_bf16.h>

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::lcf;

using base_tile = st_bf<64, 64>;
using g_layout = gl<bf16, 1, 1, -1, -1, base_tile>;
using kittens_pgl = pgl<gl<bf16, 1, 1, -1, -1, base_tile>>;

constexpr size_t N = 65536;
// constexpr size_t N = 40960;
constexpr int NUM_DEVICES = 8;

template<int M_BLOCK, int N_BLOCK>
struct matmul_layout {
    using  base_tile      = st_bf<64, 64>;
    using  global_layout  = gl<bf16, 1, 1, -1, -1, base_tile>;
    struct globals        { 
        global_layout A, B;
        kittens_pgl C_pgl; 
        int dev_idx;
    };
    struct input_block    { base_tile a[M_BLOCK], b[N_BLOCK]; };
    struct finish_block   { base_tile c[M_BLOCK][N_BLOCK]; };
    struct common_state   { int2 coord; };
    struct consumer_state { rt_fl<16, N_BLOCK*base_tile::cols> accum; }; // 16 x 256
};
template<int _M_BLOCK=2, int _N_BLOCK=4, int _SUPER_M=12>
struct matmul_template {
    static constexpr int M_BLOCK = _M_BLOCK, N_BLOCK = _N_BLOCK, SUPER_M = _SUPER_M;
    using layout    = matmul_layout<M_BLOCK, N_BLOCK>;
    using wide_tile = st_bf<64, 64*N_BLOCK>; // 64 x 256
    static constexpr int NUM_CONSUMER_WARPS=M_BLOCK*4, INPUT_PIPE_STAGES=4, PRODUCER_BARRIER_ARRIVALS=1;
    // Helper functions
    template<bool PERISISTENT_GRID=true> __host__ static inline dim3 grid(int M, int N, int K) {
        return dim3(PERISISTENT_GRID ? 132 : M*N/(M_BLOCK*N_BLOCK*layout::base_tile::num_elements));
    }
    // ThunderKittens template functions
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        int Rblocks = args.globals.C_pgl[args.globals.dev_idx].rows() / (M_BLOCK*64), Cblocks = args.globals.C_pgl[args.globals.dev_idx].cols() / (N_BLOCK*64);
        int super_rows = (Rblocks/SUPER_M)*SUPER_M,
            final_rows = Rblocks - super_rows,
            super_repeat = SUPER_M*Cblocks;
        int task_id = args.task_iter*gridDim.x + blockIdx.x;
        if (task_id < super_rows * Cblocks)
            args.common.coord = { SUPER_M*(task_id/super_repeat) + task_id%SUPER_M,
                           (task_id%super_repeat)/SUPER_M };
        else if (task_id < Rblocks*Cblocks) {
            int remainder_id = task_id - super_rows*Cblocks;
            args.common.coord = { super_rows + (remainder_id%final_rows), remainder_id/final_rows };
        }
        else { // Id is too high, no more work to do
            args.num_iters = -1;
            return;
        }
        args.num_iters = args.globals.A.cols()/64;
        int id = warpgroup::groupid() == NUM_CONSUMER_WARPS/4 ? 0 : warpgroup::groupid(); // producer sets as 0
        args.common.coord = { args.common.coord.x*M_BLOCK + id, args.common.coord.y*N_BLOCK };
    }
    struct producer {
        __device__ static void setup(producer_setup_args<layout> args) {
            warpgroup::decrease_registers<40>(); // decrease registers for producers
        }
        __device__ static void load(producer_load_args<layout> args) {
            if(warpgroup::warpid() == 0) {
                tma::expect(args.inputs_arrived, args.input);
                for(int i = 0; i < M_BLOCK; i++)
                    tma::load_async(args.input.a[i], args.globals.A,
                                    {args.common.coord.x+i, args.iter}, args.inputs_arrived);
                for(int i = 0; i < N_BLOCK; i++)
                    tma::load_async(args.input.b[i], args.globals.B,
                                    {args.iter, args.common.coord.y+i}, args.inputs_arrived);
            }
        }
    };
    struct consumer {
        __device__ static void setup(consumer_setup_args<layout> args) {
            warpgroup::increase_registers<232>(); // increase registers for consumers
            zero(args.state.accum);
        }
        __device__ static void compute(consumer_compute_args<layout> args) {
            warpgroup::mma_AB(
                args.state.accum, // dest registers
                args.input.a[warpgroup::groupid()], // A matrix
                reinterpret_cast<wide_tile&>(args.input.b) // B matrix
            );
            warpgroup::mma_async_wait();
            if(laneid() == 0) arrive(args.inputs_finished);
        }
        /*
        args.common.coord.x and y are in terms of 64 x 64 tiles 

        args.finish.c is a 2 x 4 array of 64 x 64 tiles => total data covered is 128 x 256
        
        wide tile is a 64 x 256 shared tile


        */
        __device__ static void finish(consumer_finish_args<layout> args) {
            warpgroup::store(reinterpret_cast<wide_tile&>(args.finish.c[warpgroup::groupid()]), args.state.accum);
            warpgroup::sync(warpgroup::groupid()+4);
            warpgroup::atomic_add(
                args.globals.C_pgl, 
                reinterpret_cast<wide_tile&>(args.finish.c[warpgroup::groupid()]), 
                args.globals.dev_idx, 
                {
                    args.common.coord.x,
                    args.common.coord.y / 4
                }
            );
            zero(args.state.accum);
            if (laneid() == 0) arrive(args.finish_finished);
        }
    };
};

template<typename mmt>
void inner_run(kittens::bf16 *device_A, kittens::bf16 *device_B, kittens_pgl C_pgl,
    size_t M, size_t N, size_t K, dim3 grid, dim3 block, int dev_idx) {
        
    using global_layout = typename mmt::layout::global_layout;
    using globals = typename mmt::layout::globals;

    global_layout A_global{device_A, nullptr, nullptr, M, K};
    global_layout B_global{device_B, nullptr, nullptr, K, N};
    
    globals G{A_global, B_global, C_pgl, dev_idx};

    kittens::prototype::lcf::kernel<mmt><<<grid, block, MAX_SHARED_MEMORY - 1024>>>(G);
}

// CUDA driver API
#define CUCHECK(cmd) do {                                     \
    hipError_t err = cmd;                                       \
    if (err != hipSuccess) {                                \
        const char *errStr;                                   \
        hipDrvGetErrorString(err, &errStr);                       \
        fprintf(stderr, "Failed: CUDA error %s:%d '%s'\n",    \
            __FILE__, __LINE__, errStr);                      \
        exit(EXIT_FAILURE);                                   \
    }                                                         \
} while(0)

// CUDA runtime API
#define CUDACHECK(cmd) do {                                   \
    hipError_t err = cmd;                                    \
    if (err != hipSuccess) {                                 \
        fprintf(stderr, "Failed: CUDA error %s:%d '%s'\n",    \
            __FILE__, __LINE__, hipGetErrorString(err));     \
        exit(EXIT_FAILURE);                                   \
    }                                                         \
} while(0)

void init_bf16_mat(__hip_bfloat16* matrix, size_t size, std::mt19937& prng, 
                    std::uniform_real_distribution<>& dist) {
    #pragma omp parallel for collapse(1)
    for (size_t i = 0; i < size; ++i) {
        // Convert to BF16 immediately during initialization
        matrix[i] = __float2bfloat16(dist(prng));
        
        // Print first 10 values (convert back to float for display)
        if (i < 10)
            std::cout << __bfloat162float(matrix[i]) << " ";
    }
    std::cout << "\n";
}

template<typename mmt>
void run(size_t M, size_t N, size_t K) {
    
    std::cout << "------------------------- Benchmark -------------------------\n";
    std::cout << "  M = " << M << ", N = " << N << ", K = " << K << "\n";
    std::cout << "  Block size: " << mmt::M_BLOCK * 64 << "x" << mmt::N_BLOCK * 64 << "\n";

    // Host-side matrices
    __hip_bfloat16* host_A_bf16 = new __hip_bfloat16[M * K];
    __hip_bfloat16* host_B_bf16 = new __hip_bfloat16[K * N];

    // Initialize A & B matrices
    // Random seed
    std::random_device rd;  // Hardware entropy source if available
    std::mt19937 prng(rd()); // Seed with non-deterministic value
    // std::mt19937 prng(1234);
    std::uniform_real_distribution<> random(-0.5, 0.5);
    
    std::cout << "Matrix A (M x K): ";
    // init_bf16_mat(host_A_bf16, M * K, prng, random);
    
    std::cout << "Matrix B (K x N): ";
    // init_bf16_mat(host_B_bf16, K * N, prng, random);
    
    float *host_C_ref = new float[M * N];
    // Generate expected output (just do first 10x10 tile)
    // std::cout << "  Expected C (M x N): ";
    // size_t EVAL_SIZE = N;
    // #pragma omp parallel for collapse(2)
    // for (int i = 0; i < EVAL_SIZE; i++) {
    //     for (int j = 0; j < EVAL_SIZE; j++) {
    //         float sum = 0.0f;
    //         for (int k = 0; k < K; k++) {
    //             sum += float(host_A_bf16[i * K + k]) * float(host_B_bf16[k * N + j]);
    //         }
    //         host_C_ref[i * N + j] = sum;
    //     }
    // }
    // for (int i = 0; i < 10; i++) {
    //     std::cout << host_C_ref[i] << " ";
    // }
    // std::cout << "\n";

    // Allocate device-side matrices
    size_t K_sh = K / NUM_DEVICES;
    __hip_bfloat16 *device_A_uc[NUM_DEVICES], *device_B_uc[NUM_DEVICES]; // *device_C[NUM_DEVICES]
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        CUDACHECK(hipSetDevice(dev_idx));
        CUDACHECK(hipMalloc(&device_A_uc[dev_idx], M * K_sh * sizeof(__hip_bfloat16)));
        CUDACHECK(hipMalloc(&device_B_uc[dev_idx], K_sh * N * sizeof(__hip_bfloat16)));
    }

    // Copy to device matrices
    // for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
    //     CUDACHECK(hipSetDevice(dev_idx));
    //     for (int i = 0; i < M; ++i) { // TODO: do a single hipMemcpy
    //         CUDACHECK(hipMemcpy(device_A_uc[dev_idx] + i * K_sh,      // i-th row of device A
    //                              host_A_bf16 + i * K + dev_idx * K_sh, // i-th row, dev_idx-th block of host A
    //                              K_sh * sizeof(__hip_bfloat16), 
    //                              hipMemcpyHostToDevice));
    //     }
    //     // Since B is sharded row-wise, we can do a single hipMemcpy
    //     CUDACHECK(hipMemcpy(device_B_uc[dev_idx], 
    //                          host_B_bf16 + dev_idx * K_sh * N, 
    //                          K_sh * N * sizeof(__hip_bfloat16), 
    //                          hipMemcpyHostToDevice));
    // }

    /*
        Setup multimem stuff
    */
    assert(NUM_DEVICES > 1);

    // Get device_ids
    int device_ids[NUM_DEVICES];
    for (int i = 0; i < NUM_DEVICES; ++i) device_ids[i] = i;

    size_t pgl_malloc_size = M * N * sizeof(__hip_bfloat16);

    __hip_bfloat16 **device_C_ptrs = new __hip_bfloat16*[NUM_DEVICES];
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        pglCudaMalloc<true>(NUM_DEVICES, device_ids, dev_idx, &device_C_ptrs[dev_idx], pgl_malloc_size);
    }

    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        CUDACHECK(hipSetDevice(dev_idx));
        CUDACHECK(hipMemset(device_C_ptrs[dev_idx], 0, M * N * sizeof(__hip_bfloat16)));
    }

    kittens_pgl C_pgl(device_ids, device_C_ptrs, nullptr, nullptr, M, N);


    // Initialize parallel global layout
    KittensClub club(device_ids, NUM_DEVICES); // threadpool
    unsigned long smem_size = kittens::MAX_SHARED_MEMORY - 1024; // MAX_SHARED_MEMORY = 227KB for Hopper
    club.execute([smem_size](int dev_idx) {
        CUDACHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(kittens::prototype::lcf::kernel<mmt>), 
                                       hipFuncAttributeMaxDynamicSharedMemorySize, 
                                       smem_size));
    });

    dim3 grid(mmt::grid(M, N, K_sh)); // use sharded K
    dim3 block(kittens::prototype::detail::NUM_THREADS_v<mmt>);

    constexpr int PROFILE_ITERS = 2;
    for (int i = 0; i < PROFILE_ITERS; ++i) { // warmup
        club.execute([&device_A_uc, &device_B_uc, &C_pgl, &M, &N, &K_sh, &grid, &block](int dev_idx) { // warmup
            inner_run<mmt>(device_A_uc[dev_idx], device_B_uc[dev_idx], C_pgl, M, N, K_sh, grid, block, dev_idx);
            hipDeviceSynchronize();
            // CUDACHECK(hipDeviceSynchronize());
        });
    }
    
    // Start timing
    constexpr int NUM_ITERS = 10;
    std::cout << "\n  Launching kernels with grid (" << grid.x << ", " << grid.y << "), block (" << block.x << ") on all devices\n";
    auto start = std::chrono::high_resolution_clock::now();
    // Launch!
    for (int i = 0; i < NUM_ITERS; ++i) {
        club.execute([&device_A_uc, &device_B_uc, &C_pgl, &M, &N, &K_sh, &grid, &block](int dev_idx) {
            inner_run<mmt>(device_A_uc[dev_idx], device_B_uc[dev_idx], C_pgl, M, N, K_sh, grid, block, dev_idx);
            hipDeviceSynchronize();
            // CUDACHECK(hipDeviceSynchronize());
        });
    }

    // End timing
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    double avg_seconds = elapsed.count() / NUM_ITERS;


    std::cout << "    Execution time: " << (avg_seconds * 1e3) << " ms\n";

    // Copy & convert back to host
    __hip_bfloat16 *host_C_bf16 = new __hip_bfloat16[M * N];
    float *host_C = new float[M * N];
    int random_dev_idx = 3;
    CUDACHECK(hipSetDevice(random_dev_idx));
    CUDACHECK(hipMemcpy(host_C_bf16, (void *)C_pgl[random_dev_idx].raw_ptr, M * N * sizeof(__hip_bfloat16), hipMemcpyDeviceToHost));
    // for (int i = 0; i < M * N; ++i) host_C[i] = __bfloat162float(host_C_bf16[i]);

    // std::cout << "  Matrix C (M x N): ";
    // for (int i = 0; i < 10; i++) {
    //     std::cout << host_C[i] << " ";
    // }
    // std::cout << "\n";

    // float max_error = 0.0f;
    // int error_count = 0;
    // for (int i = 0; i < M * N; ++i) {
    //     float error = std::abs(host_C[i] - host_C_ref[i]);
    //     if(error > 1.0) { // large because of bf16 vs fp32 numerics
    //         if(error_count < 20) std::cout << "Error at row " << i / N << " col " << i % N << ": " << host_C[i] << " != " << host_C_ref[i] << " (ref)" << std::endl;
    //         else if(error_count == 21) std::cout << "Too many errors to show them all.\n";
    //         error_count++;
    //     }
    //     max_error = std::max(max_error, error);
    // }
    // std::cout << "    Maximum error: " << max_error << "\n";
    // std::cout << "    Error count: " << error_count << "\n";
    // std::cout << "-------------------------------------------------------------\n";

    // Clean up
    delete[] host_A_bf16;
    delete[] host_B_bf16;
    delete[] host_C;
    delete[] host_C_bf16;
    delete[] host_C_ref;
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        CUDACHECK(hipSetDevice(dev_idx));
        CUDACHECK(hipFree(device_A_uc[dev_idx]));
        CUDACHECK(hipFree(device_B_uc[dev_idx]));
    }

    
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        pglCudaFree(dev_idx, device_C_ptrs[dev_idx], pgl_malloc_size);
    }
    pglFree(C_pgl);
}

int main() {
    run<matmul_template<2, 4, 8>>(N, N, N);
    return 0;
}