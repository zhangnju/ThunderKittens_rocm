#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "prototype.cuh"
#include <iostream>

constexpr int NUM_CONSUMERS = (2); 
constexpr int NUM_PRODUCERS = (1);

using namespace kittens;

static constexpr int Mb = 128;
static constexpr int Nb = 256;
static constexpr int Kb = 128;

struct matmul_globals {
    using a_tile = st_fp8e4m3<Mb,   Kb>;
    using b_tile = st_fp8e4m3<Nb, Kb>;
    using d_tile = st_hf<Mb, 64>;

    using a_gl = gl<fp8e4m3, 1, 1, -1, -1, a_tile>;
    using b_gl = gl<fp8e4m3, 1, 1, -1, -1, b_tile>;
    using d_gl = gl<half,    1, 1, -1, -1, d_tile>;

    a_gl a;
    b_gl b;
    d_gl d;
};

constexpr int NUM_WORKERS = (NUM_CONSUMERS + NUM_PRODUCERS) * 4;
constexpr int NUM_THREADS = NUM_WORKERS * kittens::WARP_THREADS;

__device__ static inline int get_iters_per_task(const matmul_globals &g) {
    return g.a.cols() / Kb;
}
template<int SUPER_M=8> __device__ static inline int2 get_task_idx(const matmul_globals &g, int task_iter, bool is_consumer) {
    constexpr int CLUSTER_M = 4*Mb, CLUSTER_N = Nb;
    int cluster_x = clusterIdx().x, ctarank = cluster_ctarank();
    int task_id = task_iter * (gridDim.x/2) + cluster_x;
    int Rblocks = g.d.rows() / CLUSTER_M, Cblocks = g.d.cols() / CLUSTER_N;
    int super_rows = (Rblocks/SUPER_M)*SUPER_M,
        final_rows = Rblocks - super_rows,
        super_repeat = SUPER_M*Cblocks;
    if (task_id < super_rows * Cblocks) {
        return { 
            (SUPER_M*(task_id/super_repeat) + task_id%SUPER_M)*4 + ctarank*2 + is_consumer*(warpgroup::groupid()),
            is_consumer ? (task_id%super_repeat)/SUPER_M : (task_id%super_repeat)/SUPER_M
            // is_consumer ? (task_id%super_repeat)/SUPER_M : 2*((task_id%super_repeat)/SUPER_M) + ctarank
        };
    }
    else if (task_id < Rblocks*Cblocks) {
        int remainder_id = task_id - super_rows*Cblocks;
        return {
            (super_rows + remainder_id%final_rows)*4 + ctarank*2 + is_consumer*(warpgroup::groupid()),
            is_consumer ? remainder_id/final_rows : (remainder_id/final_rows)
            // is_consumer ? remainder_id/final_rows : 2*(remainder_id/final_rows) + ctarank
        };
    }
    else {
        return { -1, -1 };
    }
}

__global__ __cluster_dims__(2) __launch_bounds__(NUM_THREADS, 1)
void matmul(const __grid_constant__ matmul_globals g) {

    extern __shared__ int __shm[]; 
    tma_swizzle_allocator al((int*)&__shm[0]);
    int warpid = kittens::warpid(), warpgroupid = warpgroup::groupid();
    int iters_per_task = get_iters_per_task(g);

    constexpr int PIPE_DEPTH = 3;

    using a_tile = matmul_globals::a_tile;
    using b_tile = matmul_globals::b_tile;
    using d_tile = matmul_globals::d_tile;
    
    a_tile (&a_smem)[PIPE_DEPTH][NUM_CONSUMERS] = al.allocate<a_tile, PIPE_DEPTH, NUM_CONSUMERS>();
    b_tile (&b_smem)[PIPE_DEPTH]                = al.allocate<b_tile, PIPE_DEPTH>();
    d_tile (&d_smem)                            = al.allocate<d_tile>();

    tensor_allocator<1, 2> tm_alloc{};
    using d_tt_t = tt<float, Mb, Nb>;

    __shared__ kittens::semaphore inputs_arrived[PIPE_DEPTH], inputs_finished[PIPE_DEPTH], outputs_arrived, outputs_finished[NUM_CONSUMERS];
    uint32_t bitfield = 0xFFFF0000; // ***_finished phase bits start as 1s, ***_arrived phase bits start as 0s

    if (threadIdx.x == 0) { 
        for(int i = 0; i < PIPE_DEPTH; i++) {
            init_semaphore(inputs_arrived[i], 0, 1); 
            init_semaphore(inputs_finished[i], 0, 2); 
        }
        init_semaphore(outputs_arrived, 0, 1);
        for(int i = 0; i < NUM_CONSUMERS; i++) {
            init_semaphore(outputs_finished[i], 0, 1);
        }
    }

    everyone::tma::cluster::sync();
    
    if(warpgroupid == NUM_CONSUMERS) {
        warpgroup::decrease_registers<56>();
        int ctarank = cluster_ctarank(); 
        if(warpgroup::warpid() == 3) {
            int input_ring = 0; // tracking which input block is being loaded
            for(int task_iter = 0; true; task_iter++) {
                int2 rowcol = get_task_idx(g, task_iter, false);
                if(rowcol.x == -1) {
                    for(int idx = 0; idx < (PIPE_DEPTH); idx++) {
                        wait(inputs_finished[input_ring], prototype::get_phasebit<1>(bitfield, input_ring));
                        input_ring=prototype::ring_advance<PIPE_DEPTH>(input_ring);
                    }
                    if(laneid() == 0) arrive(outputs_arrived);
                    break;
                }
                for (int idx = 0; idx < iters_per_task; idx++) {
                    wait(inputs_finished[input_ring], prototype::get_phasebit<1>(bitfield, input_ring));
                    prototype::update_phasebit<1>(bitfield, input_ring);
                    if(task_iter>0 && idx==PIPE_DEPTH-1 && laneid() == 0) arrive(outputs_arrived); 
                    warp::tma::expect(inputs_arrived[input_ring], a_smem[0][0], a_smem[0][1], b_smem[0]);
                    warp::tma::load_async(a_smem[input_ring][0], g.a, {(rowcol.x+0), idx}, inputs_arrived[input_ring]);
                    warp::tma::load_async(a_smem[input_ring][1], g.a, {(rowcol.x+1), idx}, inputs_arrived[input_ring]);
                    warp::tma::load_async(b_smem[input_ring],    g.b, { rowcol.y,    idx}, inputs_arrived[input_ring]);
                    input_ring=prototype::ring_advance<PIPE_DEPTH>(input_ring);
                }
            }
        }
        else if((warpgroup::warpid() == 0 || warpgroup::warpid() == 1)) { // launch the MMA's
            d_tt_t d_tt = tm_alloc.allocate<d_tt_t>(warpgroup::warpid()*Nb);
            int input_ring = 0; // tracking which input block is being loaded
            for(int task_iter = 0; true; task_iter++) {
                int2 rowcol = get_task_idx(g, task_iter, false);
                if(rowcol.x == -1) break;
                wait(outputs_finished[warpgroup::warpid()], (task_iter+1)%2); // make sure tensor memory is ready to be written to.
                wait(inputs_arrived[input_ring], prototype::get_phasebit<0>(bitfield, input_ring));
                prototype::update_phasebit<0>(bitfield, input_ring);
                warp::mm_ABt(d_tt, a_smem[input_ring][warpgroup::warpid()], b_smem[input_ring], inputs_finished[input_ring]);
                input_ring=prototype::ring_advance<PIPE_DEPTH>(input_ring);
                for(int idx = 1; idx < iters_per_task; idx++) {
                    wait(inputs_arrived[input_ring], prototype::get_phasebit<0>(bitfield, input_ring));
                    prototype::update_phasebit<0>(bitfield, input_ring);
                    warp::mma_ABt(d_tt, a_smem[input_ring][warpgroup::warpid()], b_smem[input_ring], inputs_finished[input_ring]);
                    input_ring=prototype::ring_advance<PIPE_DEPTH>(input_ring);
                }
            }
        }
    }
    else {
        warpgroup::increase_registers<224>();
        d_tt_t d_tt = tm_alloc.allocate<d_tt_t>(warpgroupid*Nb);
        for(int task_iter = 0; true; task_iter++) {
            int2 rowcol = get_task_idx(g, task_iter, true);
            if(rowcol.x == -1) break;
            kittens::wait(outputs_arrived, task_iter%2);
            rt_hf<Mb/4, d_tile::cols> d_reg[4];
            if(warpgroupid == 1) group<8>::sync(15);
            #pragma unroll
            for(int i = 0; i < Nb/d_tile::cols; i++) {
                warpgroup::load_async(d_reg[i], d_tt.subtile<tt<float, 128, 64>>(0, 64*i));
            }
            tensor_load_wait();
            warpgroup::sync(warpgroupid);
            if(warpgroup::laneid() == 0) arrive(outputs_finished[warpgroupid]); // Tensor memory for warpgroup 0 is now free.
            if(warpgroupid == 0) group<8>::sync(15);
            if(warpgroupid == 1) group<8>::sync(14);
            warpgroup::store(d_smem, d_reg[0]);
            warpgroup::sync(warpgroupid);
            if(warpgroup::warpid() == 0) warp::tma::store_async(g.d, d_smem, {rowcol.x, 4*rowcol.y+0});
            #pragma unroll
            for(int i = 1; i < Nb/d_tile::cols; i++) {
                tma::store_async_read_wait();
                warpgroup::sync(warpgroupid);
                warpgroup::store(d_smem, d_reg[i]);
                warpgroup::sync(warpgroupid);
                if(warpgroup::warpid() == 0) warp::tma::store_async(g.d, d_smem, {rowcol.x, 4*rowcol.y+i});
            }
            tma::store_async_read_wait();
            if(warpgroupid == 0) group<8>::sync(14);
            group<8>::sync(15); // All consumers sync here.
        }
    }
    everyone::tma::cluster::sync();
}