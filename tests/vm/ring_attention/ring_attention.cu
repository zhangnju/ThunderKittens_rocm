#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "vm/vm.cuh"
#include <iostream>

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::vm;

constexpr int SM_COUNT = 148;
constexpr int QO_BLOCK_SIZE = 128; // sequence length must be divisible by this * 2
constexpr int KV_BLOCK_SIZE = 128; // sequence length must be divisible by this
constexpr int HEAD_DIM = 64;

using qo_tile = st_bf<QO_BLOCK_SIZE, HEAD_DIM>;
using kv_tile = st_bf<KV_BLOCK_SIZE, HEAD_DIM>;
using a_tile = st_bf<QO_BLOCK_SIZE, KV_BLOCK_SIZE>;

using config = default_config;
struct globals {
    constexpr static int num_devices = 1;

    using instruction_layout = ::kittens::prototype::vm::instruction_layout<config>;
    using barrier_layout = gl<uint, 1, 1, 1, num_devices>;
    using timing_layout = ::kittens::prototype::vm::timing_layout<config>;
    
    using qo_layout = gl<bf16, -1, -1, -1, HEAD_DIM, qo_tile>; // Batch, Head, Seq, Dim (full MHA)
    using kv_layout = gl<bf16, -1, -1, -1, HEAD_DIM, kv_tile>;

    instruction_layout instructions;
    gl_array<barrier_layout, num_devices> barriers;
    timing_layout timings;

    qo_layout Q; // local Q sharded on sequence dimension
    gl_array<kv_layout, num_devices> K0s;
    gl_array<kv_layout, num_devices> K1s;
    gl_array<kv_layout, num_devices> V0s;
    gl_array<kv_layout, num_devices> V1s;
    qo_layout O;

    dim3 grid() { return dim3(SM_COUNT); }
    dim3 block() { return dim3(config::NUM_THREADS); }
    int dynamic_shared_memory() { return config::DYNAMIC_SHARED_MEMORY; }
};

template<typename config=config> struct RingAttentionOp {
    static constexpr int opcode = 725;
    static constexpr int PIPELINE_STAGES = 2;
    static_assert(config::NUM_CONSUMER_WARPS == 8, "RingAttentionOp only supports 2 consumer warpgroups");

    struct parsed_instruction {
        int B;             // batch index              (in units of 1)
        int H;             // head index               (in units of 1)
        int QO_idx;        // local Q block index      (in units of `QO_BLOCK_SIZE * 2` tokens)
        int num_kv_blocks; // # of KV blocks to handle (in units of `KV_BLOCK_SIZE` tokens)
        int ring_stage;    // current ring stage index (0, 1, ..., NUM_DEVS - 1)
        int num_comms;     // number of SMs doing comms
        int num_comps;     // number of instructions per ring stage per device
        int dev_idx;       // current device index     (0, 1, ..., NUM_DEVS - 1)
        __device__ inline parsed_instruction(typename config::instruction_t &instruction) {
            B = instruction[1];
            H = instruction[2];
            QO_idx = instruction[3];
            num_kv_blocks = instruction[4];
            ring_stage = instruction[5];
            num_comms = instruction[6];
            num_comps = instruction[7];
            dev_idx = instruction[8];
        }
        __device__ inline parsed_instruction(state<config> &s): parsed_instruction(s.instruction()) {}
    };

    __device__ static inline semaphore &q_arrived(state<config> &s, int id)      { return s.semaphores()[id]; }
    __device__ static inline semaphore &o_arrived(state<config> &s, int id)      { return s.semaphores()[2 + id]; }
    __device__ static inline semaphore &qk_unloaded(state<config> &s, int id)    { return s.semaphores()[4 + id]; }
    __device__ static inline semaphore &av_ready(state<config> &s, int id)       { return s.semaphores()[6 + id]; }
    __device__ static inline semaphore &k_arrived(state<config> &s, int stage)   { return s.semaphores()[8 + PIPELINE_STAGES * 0 + stage]; }
    __device__ static inline semaphore &v_arrived(state<config> &s, int stage)   { return s.semaphores()[8 + PIPELINE_STAGES * 1 + stage]; }
    __device__ static inline semaphore &qk_finished(state<config> &s, int stage) { return s.semaphores()[8 + PIPELINE_STAGES * 2 + stage]; }
    __device__ static inline semaphore &av_finished(state<config> &s, int stage) { return s.semaphores()[8 + PIPELINE_STAGES * 3 + stage]; }

    __device__ static inline int get_q_page(state<config> &s, int id)    { return id; } // use PIDs for now
    __device__ static inline int get_a_page(state<config> &s, int id)    { return 2 + id; }
    __device__ static inline int get_o_page(state<config> &s, int id)    { return 4 + id; }
    __device__ static inline int get_k_page(state<config> &s, int stage) { return 6 + PIPELINE_STAGES * 0 + stage; }
    __device__ static inline int get_v_page(state<config> &s, int stage) { return 6 + PIPELINE_STAGES * 1 + stage; }

    struct controller {
        static __device__ int release_lid(const globals &g, typename config::instruction_t &instruction, int &query) {
            int lids[config::NUM_PAGES] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12};
            return lids[query];
        }
        static __device__ int init_semaphores(const globals &g, state<config> &s) {
            for (int i = 0; i < 2; ++i) {
                init_semaphore(q_arrived(s, i), 1);
                init_semaphore(o_arrived(s, i), 4);
                init_semaphore(qk_unloaded(s, i), 4);
                init_semaphore(av_ready(s, i), 4);
            }
            for (int i = 0; i < PIPELINE_STAGES; ++i) {
                init_semaphore(k_arrived(s, i), 1);
                init_semaphore(v_arrived(s, i), 1);
                init_semaphore(qk_finished(s, i), 2);
                init_semaphore(av_finished(s, i), 2);
            }
            return 2*4 + PIPELINE_STAGES*4;
        }
    };

    struct loader {
        static __device__ void run(const globals &g, state<config> &s) {
            // TODO: wait for comms complete
            parsed_instruction inst{s};
            int laneid = warp::laneid();
            if (laneid < 2) { // Load Q for the 2 consumer warpgroups
                auto q_page = get_q_page(s, laneid);
                auto &q = *reinterpret_cast<qo_tile *>(s.pages[q_page].data);
                s.wait_page_ready(q_page);
                tma::expect(q_arrived(s, laneid), q);
                tma::load_async(q, g.Q, {inst.B, inst.H, inst.QO_idx + laneid, 0}, q_arrived(s, laneid));
                // printf("Q load start %d\n", laneid);
            } else if (laneid == 2) { // Load Ks
                uint32_t phasebit = 0;
                for (int i = 0; i < inst.num_kv_blocks; i++) {
                    int stage = i % PIPELINE_STAGES;
                    auto k_page = get_k_page(s, stage);
                    auto &k = *reinterpret_cast<kv_tile *>(s.pages[k_page].data);
                    if (i < PIPELINE_STAGES) {
                        s.wait_page_ready(get_k_page(s, stage));
                    } else {
                        wait(qk_finished(s, stage), get_phasebit<0>(phasebit, stage));
                        update_phasebit<0>(phasebit, stage);
                    }
                    tma::expect(k_arrived(s, stage), k);
                    // Todo: vary by ring stage
                    tma::load_async(k, g.K0s[inst.dev_idx], {inst.B, inst.H, i, 0}, k_arrived(s, stage));
                    // printf("K load start %d\n", i);
                }
                for (int i = 0; i < PIPELINE_STAGES; i++) {
                    int stage = (i + inst.num_kv_blocks) % PIPELINE_STAGES;
                    wait(qk_finished(s, stage), get_phasebit<0>(phasebit, stage));
                    // printf("arriving K finished page %d\n", stage);
                    s.finish_page(get_k_page(s, stage), config::NUM_CONSUMER_WARPS);
                    update_phasebit<0>(phasebit, stage);
                }
            } else if (laneid == 3) { // Load Vs
                uint32_t phasebit = 0;
                for (int i = 0; i < inst.num_kv_blocks; i++) {
                    int stage = i % PIPELINE_STAGES;
                    auto v_page = get_v_page(s, stage);
                    auto &v = *reinterpret_cast<kv_tile *>(s.pages[v_page].data);
                    if (i < PIPELINE_STAGES) {
                        s.wait_page_ready(get_v_page(s, stage));
                    } else {
                        wait(av_finished(s, stage), get_phasebit<0>(phasebit, stage));
                        update_phasebit<0>(phasebit, stage);
                    }
                    tma::expect(v_arrived(s, stage), v);
                    tma::load_async(v, g.V0s[inst.dev_idx], {inst.B, inst.H, i, 0}, v_arrived(s, stage));
                    // printf("V load start %d\n", i);
                }
                for (int i = 0; i < PIPELINE_STAGES; i++) {
                    int stage = (i + inst.num_kv_blocks) % PIPELINE_STAGES;
                    wait(av_finished(s, stage), get_phasebit<0>(phasebit, stage));
                    s.finish_page(get_v_page(s, stage), config::NUM_CONSUMER_WARPS);
                    update_phasebit<0>(phasebit, stage);
                }
            } else if (6 + PIPELINE_STAGES*2 <= laneid && laneid < config::NUM_PAGES) { // Finish unused pages
                s.wait_page_ready(laneid);
                s.finish_page(laneid, config::NUM_CONSUMER_WARPS);
                // printf("page %d finished\n", laneid);
            }
        }
    };

    struct launcher {
        static __device__ void run(const globals &g, state<config> &s) {
            parsed_instruction inst{s};
            int laneid = warp::laneid();

            // Nothing is ready until the tensor cores are ready
            s.wait_tensor_ready();

            if (laneid < 2) { // Launch Q @ K^T for the 2 consumer warpgroups
                auto q_page = get_q_page(s, laneid);
                auto &q = *reinterpret_cast<qo_tile *>(s.pages[q_page].data);
                wait(q_arrived(s, laneid), 0);
                // printf("Q load done %d\n", laneid);
                
                uint32_t phasebit = 0;
                for (int i = 0; i < inst.num_kv_blocks; ++i) {
                    int stage = i % PIPELINE_STAGES;
                    if (i > 0) {
                        wait(qk_unloaded(s, laneid), get_phasebit<1>(phasebit, laneid));
                        update_phasebit<1>(phasebit, laneid);
                        // printf("QK unload done %d - %d\n", laneid, i - 1);
                    }
                    auto k_page = get_k_page(s, stage);
                    auto &k = *reinterpret_cast<kv_tile *>(s.pages[k_page].data);
                    wait(k_arrived(s, stage), get_phasebit<0>(phasebit, stage));
                    // printf("K load done %d\n", i);
                    // if (laneid == 0) {
                    //     printf("Launching QK %d - %d\n", laneid, i);
                    //     printf("Q:");
                    //     for (int x = 0; x < 128; x++)
                    //         printf("%f ", float(q[x]));
                    //     printf("\n");
                    //     printf("K:");
                    //     for (int x = 0; x < 128; x++)
                    //         printf("%f ", float(k[x]));
                    //     printf("\n");
                    // }


                    auto qk_accumulator = s.tensor_alloc.template allocate<tt<float, QO_BLOCK_SIZE, KV_BLOCK_SIZE>>(laneid*KV_BLOCK_SIZE);
                    mm_ABt(qk_accumulator, q, k, qk_finished(s, stage));
                    // printf("qk launched %d - %d\n", laneid, i);
                    update_phasebit<0>(phasebit, stage);
                }
            } else if (laneid < 4) { // Launch ATT @ V for the 2 consumer warpgroups
                auto att_page = get_a_page(s, laneid-2);
                auto &att = *reinterpret_cast<a_tile *>(s.pages[att_page].data);

                uint32_t phasebit = 0;
                for (int i = 0; i < inst.num_kv_blocks; ++i) {
                    int stage = i % PIPELINE_STAGES;
                    auto v_page = get_v_page(s, stage);
                    auto &v = *reinterpret_cast<kv_tile *>(s.pages[v_page].data);
                    wait(v_arrived(s, stage), get_phasebit<0>(phasebit, stage));
                    wait(av_ready(s, laneid-2), get_phasebit<1>(phasebit, laneid-2));
                    
                    // printf("v load done and av ready %d - %d\n", laneid, i);
                    auto av_accumulator = s.tensor_alloc.template allocate<tt<float, QO_BLOCK_SIZE, HEAD_DIM>>(2*KV_BLOCK_SIZE+(laneid-2)*HEAD_DIM);
                    mma_AB(av_accumulator, att, v, av_finished(s, stage));
                    update_phasebit<0>(phasebit, stage);
                    update_phasebit<1>(phasebit, laneid-2);
                    // printf("av launched %d - %d\n", laneid, i);
                }
            }
        }
    };

    struct consumer {
        static __device__ void run(const globals &g, state<config> &s) {                   
            parsed_instruction inst{s};
            int warpid = warpgroup::warpid();
            int groupid = warpgroup::groupid();

            // constexpr float softmax_scale = 0.08838834764831843f;         // 1 / sqrt(HEAD_DIM=128)
            constexpr float softmax_scale = 0.125;                         // 1 / sqrt(HEAD_DIM=64)
            constexpr float softmax_temp = softmax_scale * 1.44269504089f; // 1 / {sqrt(HEAD_DIM=128) * ln(2)}

            rt_fl<QO_BLOCK_SIZE / 4, KV_BLOCK_SIZE> att_fl;
            rt_fl<QO_BLOCK_SIZE / 4, HEAD_DIM> out_fl;
            col_vec<rt_fl<QO_BLOCK_SIZE / 4, KV_BLOCK_SIZE>> max_vec;
            col_vec<rt_fl<QO_BLOCK_SIZE / 4, KV_BLOCK_SIZE>> scaled_max_vec;
            col_vec<rt_fl<QO_BLOCK_SIZE / 4, KV_BLOCK_SIZE>> last_scaled_max_vec;
            col_vec<rt_fl<QO_BLOCK_SIZE / 4, KV_BLOCK_SIZE>> diff_scaled_max_vec;
            col_vec<rt_fl<QO_BLOCK_SIZE / 4, HEAD_DIM>> norm_vec;

            warp::zero(out_fl);
            warp::neg_infty(max_vec);
            warp::zero(last_scaled_max_vec); // just not +-inf
            warp::zero(norm_vec);

            auto qk_accumulator = s.tensor_alloc.template allocate<tt<float, QO_BLOCK_SIZE, KV_BLOCK_SIZE>>(groupid*KV_BLOCK_SIZE);
            auto av_accumulator = s.tensor_alloc.template allocate<tt<float, QO_BLOCK_SIZE, HEAD_DIM>>(2*KV_BLOCK_SIZE + groupid*HEAD_DIM);

            uint32_t phasebit = 0;
            for (int i = 0; i < inst.num_kv_blocks; ++i) {
                int stage = i % PIPELINE_STAGES;

                // Read in QK^T
                wait(qk_finished(s, stage), get_phasebit<0>(phasebit, stage)); // wait for mm to finish
                // printf("QK finished %d - %d\n", groupid, i);
                if (i == inst.num_kv_blocks - 1) {
                    s.warp_finish_page(get_q_page(s, groupid), config::NUM_CONSUMER_WARPS / 4);
                }
                warpgroup::load_async(att_fl, qk_accumulator);
                tensor_load_wait();
                __syncwarp();
                warp::arrive(qk_unloaded(s, groupid));


                auto &test = *reinterpret_cast<st_bf<QO_BLOCK_SIZE, KV_BLOCK_SIZE> *>(s.pages[10].data);
                warpgroup::store(test, att_fl);
                for (int x = 0; x < 100; x++) {
                    if (groupid == 0 && warpgroup::laneid() == 0)
                         printf("%f ", float(test[x]));
                }
                if (groupid == 0 && warpgroup::laneid() == 0)
                    printf("\n");



                // Get maximums and scale by softmax temp
                warp::row_max(max_vec, att_fl, max_vec);
                warp::mul(att_fl, att_fl, softmax_temp);
                warp::mul(scaled_max_vec, max_vec, softmax_temp);

                // Compute softmax numerator
                warp::sub_row(att_fl, att_fl, scaled_max_vec);
                warp::exp2(att_fl, att_fl);

                // Compute normalizer
                warp::sub(diff_scaled_max_vec, last_scaled_max_vec, scaled_max_vec);
                warp::exp2(diff_scaled_max_vec, diff_scaled_max_vec);
                warp::copy(last_scaled_max_vec, scaled_max_vec); // save for next iteration

                // Prepare for AV
                auto att_page = get_a_page(s, groupid);
                auto &att = *reinterpret_cast<a_tile *>(s.pages[att_page].data);
                if (i == 0) {
                    s.wait_page_ready(att_page);
                } else {
                    int prev_stage = (i + PIPELINE_STAGES - 1) % PIPELINE_STAGES;
                    // printf("waiting for av finished %d - %d\n", groupid, prev_stage);
                    wait(av_finished(s, prev_stage), get_phasebit<0>(phasebit, prev_stage)^1); // wait for the previous mma to finish
                    // printf("av finished %d - %d\n", groupid, prev_stage);
                    warpgroup::load_async(out_fl, av_accumulator);
                    tensor_load_wait(); // TODO: is this needed?
                }
                warp::mul_row(out_fl, out_fl, diff_scaled_max_vec); // normalize previous outputs
                warpgroup::store_async(av_accumulator, out_fl);
                warpgroup::store(att, att_fl);
                tensor_store_wait();
                __syncwarp();
                warp::arrive(av_ready(s, groupid));

                // Normalize and accumulate softmax denominator
                warp::mul(norm_vec, norm_vec, diff_scaled_max_vec);
                warp::row_sum(norm_vec, att_fl, norm_vec);

                update_phasebit<0>(phasebit, stage);
            }

            // printf("Arrivedddddd\n");

            // Finish
            int last_stage = (inst.num_kv_blocks - 1) % PIPELINE_STAGES;
            wait(av_finished(s, last_stage), get_phasebit<0>(phasebit, last_stage)^1);
            warpgroup::load_async(out_fl, av_accumulator);
            s.warp_finish_page(get_a_page(s, groupid), config::NUM_CONSUMER_WARPS / 4);
            tensor_load_wait();
            warp::arrive(s.tensor_finished);
            warp::div_row(out_fl, out_fl, norm_vec);

            // printf("Arrivedddddd2\n");

            int out_page = get_o_page(s, groupid);
            auto &out = *reinterpret_cast<qo_tile *>(s.pages[out_page].data);
            s.wait_page_ready(out_page);
            warpgroup::store(out, out_fl);
            warp::arrive(o_arrived(s, groupid));

            // printf("Arrivedddddd3\n");
        }
    };
    struct storer {
        static __device__ void run(const globals &g, state<config> &s) {
            parsed_instruction inst{s};
            int laneid = warp::laneid();
            if (laneid < 2) {

                // printf("Arrivedddddd4\n");
                int out_page = get_o_page(s, laneid);
                auto &out = *reinterpret_cast<qo_tile *>(s.pages[out_page].data);
                wait(o_arrived(s, laneid), 0);

                // printf("Arrivedddddd5\n");
                tma::store_async(g.O, out, {inst.B, inst.H, inst.QO_idx + laneid, 0});
                tma::store_async_read_wait(); // or wait until read complete

                // printf("Arrivedddddd6\n");
                s.finish_page(out_page, config::NUM_CONSUMER_WARPS);
            }
            // TODO: write to barrier
        }
    };
};

#include "pyutils/pyutils.cuh"

PYBIND11_MODULE(ring_attention, m) {
    m.doc() = "ring attention python module";
    kittens::py::bind_kernel<kvm<config, globals,
        RingAttentionOp<config>
    >>(m, "ring_attention",
        &globals::instructions,
        &globals::barriers,
        &globals::timings,
        &globals::Q,
        &globals::K0s,
        &globals::K1s,
        &globals::V0s,
        &globals::V1s,
        &globals::O
    );
}
